#include <hip/hip_runtime.h>
#include "ffi.h"
#include "gemm.cu"

#define BuildGPUMul(bits)        \
struct GPUMul##bits {            \
    GPUMul<uint##bits##_t> *ctx; \
};                               \
\
GPUMul##bits##_t *mul_new_##bits(uint64_t m, uint64_t k, uint64_t n) {  \
    auto *ctx = (GPUMul##bits##_t*) malloc(sizeof(GPUMul##bits##_t));   \
    ctx->ctx = new GPUMul<uint##bits##_t>(m, k, n);                     \
    return ctx;                                                         \
}                                                                       \
\
void mul_free_##bits(GPUMul##bits##_t *ctx) { \
    delete ctx->ctx;                          \
    free(ctx);                                \
}                                             \
\
void allocate_##bits(GPUMul##bits##_t *ctx, uint64_t m, uint64_t k, uint64_t n) { \
    ctx->ctx->allocate(m, k, n);                                                  \
}                                                                                 \
\
void set_batch_##bits(GPUMul##bits##_t *ctx, uint64_t n) { \
    ctx->ctx->set_batch(n);                                \
}                                                          \
\
uint32_t* get_host_a_##bits(GPUMul##bits##_t *ctx) { \
    return ctx->ctx->raw_host_a();                   \
}                                                    \
\
uint##bits##_t* get_host_data_##bits(GPUMul##bits##_t *ctx, int index) { \
    return ctx->ctx->raw_host_data(index);                               \
}                                                                        \
\
void sync_device_##bits(GPUMul##bits##_t *ctx, int index) { \
    ctx->ctx->sync_device(index);                           \
}                                                           \
\
void gemm_##bits(GPUMul##bits##_t *ctx) {                             \
    ctx->ctx->gemm();                                                 \
}                                                                     \

bool use_gpu() {
    int deviceCount;
    hipError_t e = hipGetDeviceCount(&deviceCount);
    return e == hipSuccess ? deviceCount > 0 : -1;
}

BuildGPUMul(32);
BuildGPUMul(64);
