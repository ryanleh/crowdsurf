#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "utils.cuh"

template<typename T>
class GPUMul {
public:
    using Rows = cutlass::layout::RowMajor;
    using Cols = cutlass::layout::ColumnMajor;
    using Gemm = cutlass::gemm::device::Gemm<uint32_t, Rows, T, Cols, T, Rows>;

private:
    // The plaintext space is always < 32-bits for the parameters we consider
    cutlass::HostTensor<uint32_t, Rows> a_;
    // This is stored in column-major format so that when we modulus
    // switch a batch of queries, we can write them directly into host
    // memory without fragmented caching
    cutlass::HostTensor<T, Cols> b_;
    cutlass::HostTensor<T, Rows> c_;
    cutlass::HostTensor<T, Rows> d_;

    cutlass::device_memory::allocation<uint8_t> workspace_;
    Gemm gemm_op_;
    T alpha_ = T(1);
    T beta_ = T(1);
    int split_k_slices_ = 1;
    uint64_t _m, _k;

public:
    GPUMul() {
        // Check that our GPU supports the necessary operations
        hipDeviceProp_t props;
        CUDA_CHECK(hipGetDeviceProperties(&props, 0));
        if (props.major != 7) {
            std::cerr << "GPU must have compute capability >70" << std::endl;
            exit(1);
        }
    }

    GPUMul(uint64_t m, uint64_t k, uint64_t n): GPUMul() {
        allocate(m, k, n);
    }

    void allocate(uint64_t m, uint64_t k, uint64_t n) {
        _m = m;
        _k = k;
        cutlass::gemm::GemmCoord problem_size(m, n, k);
        
        // Allocate memory for each tensor
        a_.reset(problem_size.mk());
        b_.reset(problem_size.kn());
        c_.reset(problem_size.mn());
        d_.reset(problem_size.mn());

        // Zero-out tensor data
        cutlass::reference::host::TensorFill(a_.host_view());
        cutlass::reference::host::TensorFill(b_.host_view());
        cutlass::reference::host::TensorFill(c_.host_view());
        cutlass::reference::host::TensorFill(d_.host_view());
        for (int i = 0; i < 4; i++)
            sync_device(i);
                
        // Create a tuple of GEMM kernel arguments
        typename Gemm::Arguments arguments{
            problem_size,
            a_.device_ref(),
            b_.device_ref(),
            c_.device_ref(),
            d_.device_ref(),
            {alpha_, beta_},
            split_k_slices_
        };

        // Allocate necessary workspace on device
        int workspace_size = Gemm::get_workspace_size(arguments);
        workspace_ = cutlass::device_memory::allocation<uint8_t>(workspace_size);

        // Instantiate CUTLASS kernel
        cutlass::Status status = gemm_op_.can_implement(arguments);
        CUTLASS_CHECK(status);

        status = gemm_op_.initialize(arguments, workspace_.get());
        CUTLASS_CHECK(status);
    }

    void set_batch(uint64_t n) {
        cutlass::gemm::GemmCoord problem_size(_m, n, _k);
        
        // Allocate memory for each tensor
        b_.reset(problem_size.kn());
        c_.reset(problem_size.mn());
        d_.reset(problem_size.mn());

        // Zero-out tensor data
        cutlass::reference::host::TensorFill(b_.host_view());
        cutlass::reference::host::TensorFill(c_.host_view());
        cutlass::reference::host::TensorFill(d_.host_view());
        for (int i = 1; i < 4; i++)
            sync_device(i);
                
        // Create a tuple of GEMM kernel arguments
        typename Gemm::Arguments arguments{
            problem_size,
            a_.device_ref(),
            b_.device_ref(),
            c_.device_ref(),
            d_.device_ref(),
            {alpha_, beta_},
            split_k_slices_
        };

        // Allocate necessary workspace on device
        int workspace_size = Gemm::get_workspace_size(arguments);
        workspace_ = cutlass::device_memory::allocation<uint8_t>(workspace_size);

        // Instantiate CUTLASS kernel
        cutlass::Status status = gemm_op_.can_implement(arguments);
        CUTLASS_CHECK(status);

        status = gemm_op_.initialize(arguments, workspace_.get());
        CUTLASS_CHECK(status);
    }


    uint32_t* raw_host_a() {
        return a_.host_data();
    }

    T* raw_host_data(int index) {
        switch (index) {
            case 1:
                return b_.host_data();
            case 2:
                return c_.host_data();
            case 3:
                return d_.host_data();
        }
        return nullptr;
    }

    void sync_device(int index) {
        switch (index) {
            case 0:
                a_.sync_device();
                break;
            case 1:
                b_.sync_device();
                break;
            case 2:
                c_.sync_device();
                break;
            case 3:
                d_.sync_device();
                break;
        }
    }

    void gemm() {
        // Execute kernel
        auto status = gemm_op_();
        CUTLASS_CHECK(status);
      
        // wait for kernels to finish
        hipDeviceSynchronize();
      
        // Sync host with result device data
        d_.sync_host();
    }
};
